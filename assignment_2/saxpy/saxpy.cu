#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;
    int size = totalBytes/3;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //

    hipMalloc(&device_x,size);
    hipMalloc(&device_y,size);
    hipMalloc(&device_result,size);


    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //

    hipMemcpy(device_x,xarray,size,hipMemcpyHostToDevice);
    hipMemcpy(device_y,yarray,size,hipMemcpyHostToDevice);
    //hipMemcpy(device_result,resultarray,size,hipMemcpyHostToDevice);

    double runStartTime = CycleTimer::currentSeconds();

    // run kernel
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();

    double runEndTime = CycleTimer::currentSeconds();

    //
    // TODO copy result from GPU using hipMemcpy
    //

    hipMemcpy(resultarray,device_result,size,hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double runDuration = runEndTime - runStartTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Just Run: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * runDuration, toBW(totalBytes, runDuration));

    // TODO free memory buffers on the GPU

    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
